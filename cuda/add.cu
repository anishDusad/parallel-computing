
#include <hip/hip_runtime.h>
 __global__ void add(int *a, int *b, int *c) {
   *c = *a + *b;
 }

int main(void) {
   int a, b, c;
   int *d_a, *d_b, *d_c;
   int size = sizeof(int);
   // host copies of a, b, c
   // device copies of a, b, c
   // Allocate space for device copies of a, b, c
   hipMalloc((void **)&d_a, size);
   hipMalloc((void **)&d_b, size);
   hipMalloc((void **)&d_c, size);
   // Setup input values
   a = 2;
   b = 7;
  // Copy inputs to device
   hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
   hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
   // Launch add() kernel on GPU
   add<<<1,1>>>(d_a, d_b, d_c);
   // Copy result back to host
   hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
   // Cleanup
   hipFree(d_a); hipFree(d_b); hipFree(d_c);
   return 0;
 }
